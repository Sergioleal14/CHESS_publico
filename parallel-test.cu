#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>
#include "definiciones.h"
#include "busqueda.cuh"
#define N 10
//
#define FENP "rnbqkb1r/pp1p1pPp/8/2p1pP2/1P1P4/3P3P/P1P5/RNBQKBNR w KQkq e6 0 1"

int main(void) {
  struct timeval inicio, fin;
  TABLERO *tab=NULL,*d_tab;
  int count1, count2,*acc_counts;
  MOVE *jugada1, *jugada2;
  int bestScore;
  MOVE **Best;
  INFO info;
  info.tiempo = 0;
  info.maxtemp = 1000000000;
	info.visited=0;
	info.stop=FALSE;
  info.depth = 2;
  long time;

  int count;

  Best=(MOVE**)malloc(sizeof(MOVE*));

  tab = Create_tablero();
  LeerFen(START_FEN,tab);




  for(int i = 0; i < N;i++){
    gettimeofday(&inicio, NULL);
    bestScore = AlphaBeta(-50000,50000,2,tab,&info,Best,NULL,0,0);
    gettimeofday(&fin, NULL);
    time=  (fin.tv_sec - inicio.tv_sec) * 1000000L + (fin.tv_usec - inicio.tv_usec);
    printf("%ld, \n",time);
    hipFree(d_tab);
  }

  return 0;
}